#include "funciones.h"  // archivo auxiliar con las funciones a utilizar.

int main(){
    matriz h_A, h_B, h_resultado; // variables tipo matriz que se ocupara en el host
    matriz dev_A, dev_B, dev_resultado; // variables tipo matriz que se ocupara en el device
	int mA, nA, mB, nB; // variables enteras para los tamanos de las matrices.
    
    cout<<"Bienvenido al programa para operar matrices."<<endl; // Solicitud de los tamanos de la matriz A.
    cout<<"Ingresa el tamano de la primera matriz (A), ejemplo, " << endl;
    cout<<"para crear una matriz 5x7 ingresa 5 7 "<<endl;
    cin >> nA >> mA;
	
	h_A.n = nA; // asignamos los tamanos a la struct matriz.  
	h_A.m = mA;
    h_A.df = new int [nA*mA]; // apartamos la memoria dinamica.
	h_A = generar_matriz_aleatoria(h_A); // generamos numeros aleatorias para la matriz.
	cout << " La matriz A es:" << endl;
	imprimir_matriz(h_A); // imprimimos la matriz 
	
	cout<<"Ingresa el tamano de la segunda matriz (B), ejemplo, "<<endl;  // solicitud de los tamanos de la matriz B.
    cout << "para crear una matriz 5x7 ingresa 5 7 " << endl;
    cin >> nB >> mB; // generamos la matriz B.
    
    h_B.n = nB; // asignamos los tamanos a la struct matriz. 
    h_B.m = mB;
    h_B.df = new int [nB*mB]; // apartamos la memoria dinamica.
    h_B = generar_matriz_aleatoria(h_B); // generamos numeros aleatorias para la matriz.
    cout << " La matriz B es:" << endl;
    imprimir_matriz(h_B); // imprimimos la matriz
    
    dev_A.n = h_A.n; // asignamos los tamanos a la struct matriz A que estara en device
    dev_A.m = h_A.m;     
    hipMalloc(&dev_A.df, nA*mA*sizeof(int)); // asignacion de memoria lineal en device
    hipMemcpy(dev_A.df, h_A.df, nA*mA*sizeof(int), hipMemcpyHostToDevice); // copiamos los datos del host al device 

    dev_B.n = h_B.n; // asignamos los tamanos a la struct matriz B que estara en device
    dev_B.m = h_B.m;
    hipMalloc(&dev_B.df, nB*mB*sizeof(int)); // asignacion de memoria lineal en device
	hipMemcpy(dev_B.df, h_B.df, nB*mB*sizeof(int), hipMemcpyHostToDevice); // copiamos los datos del host al device 

    char x, operacion, y; // variables para realizar operaciones
    cout << "Haz creado la matriz A y B. Ahora ingresa la operacion a realizar,ejemplos:" << endl;
    cout << "A+B. |A-B. |A*B. |B*A. (Nota: con el punto)" << endl; // solicitamos la operacion
    cin >> x >> operacion >> y;
    
    if (x=='A' && y=='B' && operacion=='*'){
        h_resultado.n = dev_A.n;  // asignamos los tamanos a la struct matriz para los resultados de la multiplicacion en host
        h_resultado.m = dev_B.m;
        h_resultado.df = new int [dev_A.n*dev_B.m]; // apartamos la memoria dinamica para los resultados.
        
        dev_resultado.n = h_resultado.n; // asignamos los tamanos a la struct matriz para los resultados de la multiplicacion en device
        dev_resultado.m = h_resultado.m;
        hipMalloc(&dev_resultado.df, dev_A.n*dev_B.m*sizeof(int)); // asignacion de memoria lineal en device
    }
    else if (x=='B' && y=='A' && operacion=='*'){
        h_resultado.n = dev_B.n; // asignamos los tamanos a la struct matriz para los resultados de la multiplicacion en host
        h_resultado.m = dev_A.m;
        h_resultado.df = new int [dev_B.n*dev_A.m]; // apartamos la memoria dinamica para los resultados.
        
        dev_resultado.n = h_resultado.n; // asignamos los tamanos a la struct matriz para los resultados de la multiplicacion en device
        dev_resultado.m = h_resultado.m;
        hipMalloc(&dev_resultado.df, dev_B.n*dev_A.m*sizeof(int)); // asignacion de memoria lineal en device
    }
    else{
        h_resultado.n = dev_A.n; // asignamos los tamanos a la struct matriz para los resultados de la suma/resta en host
        h_resultado.m = dev_A.m;
        h_resultado.df = new int [dev_A.n*dev_A.m]; // apartamos la memoria dinamica para los resultados.
        
        dev_resultado.n = h_resultado.n; // asignamos los tamanos a la struct matriz para los resultados de la suma/resta en device
        dev_resultado.m = h_resultado.m;
        hipMalloc(&dev_resultado.df, dev_A.n*dev_A.m*sizeof(int)); // asignacion de memoria lineal en device
    }
    
    ejecutar_operacion(dev_A, dev_B, operacion, dev_resultado, h_resultado, x, y); // ejecutamos la operacion
    
	
	return 0;
}
