
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>


__global__ void reverse_vect(int *arr, int siz){

	extern __shared__ int s[];
	int t = threadIdx.x;
	int tr = siz-t-1;

	s[t] = arr[t];
	__syncthreads();
	arr[t]=s[tr];

}


int main(){
	int n=10; 
	int *arr,*darr;
	arr = new int [n];
	hipMalloc((void**) &darr,n*sizeof(int));
	for(int i=0;i<n;i++){
		arr[i] = rand()%10;
		printf("%d\t",arr[i]);
	}
	printf("\n");
	hipMemcpy(darr,arr,n*sizeof(int),hipMemcpyHostToDevice);

	reverse_vect<<<1,n,n*sizeof(int)>>>(darr,n);

	hipMemcpy(arr,darr,n*sizeof(int),hipMemcpyDeviceToHost);
	for(int i=0;i<n;i++){
		printf("%d \t ", arr[i]);
	}
	printf("\n");

	delete arr;
	hipFree(darr);
	return 0;
}
