
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


#define MAXTHREADS 256

__global__ void suma_vect(int *a, int *b, int *c,int n){
	int i= threadIdx.x;
	c[i]=a[i]+b[i];
}

__global__ void suma_vect2(int *a, int *b, int *c,int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if(i<n){
		c[i] = a[i] + b[i];
	}
}

int main(){
	int *a,*b,*c;
	int *dev_a,*dev_b,*dev_c;
	
	int n=10;

	a=new int[n];
	b=new int[n];
	c=new int[n];

 	hipError_t cudastatus;

	cudastatus = hipMalloc((void**) &dev_a, n*sizeof(int));

	if(cudastatus != hipSuccess){
		printf("Error\n");
		exit(0);
	}
	hipMalloc((void**) &dev_b,n*sizeof(int));
	hipMalloc((void**) &dev_c,n*sizeof(int));

	for(int i=0; i<n; i++){
		a[i]=rand()%10;
		b[i]=rand()%10;
	}


	cudastatus = hipMemcpy(dev_a,a,n*sizeof(int), hipMemcpyHostToDevice);

	if(cudastatus!=hipSuccess){
		printf("Error \n");
		exit(0);
	}

	cudastatus = hipMemcpy(dev_b,b,n*sizeof(int), hipMemcpyHostToDevice);

	if(cudastatus != hipSuccess){
		printf("Error\n");
		exit(0);
	}
	//Ejecución del código sobre GPU
int nBlock = (int)ceil((double)n/MAXTHREADS);
	suma_vect<<<nBlock,MAXTHREADS>>>(dev_a,dev_b,dev_c,n);

	cudastatus = hipMemcpy(c,dev_c,n*sizeof(int),hipMemcpyDeviceToHost );

	if(cudastatus != hipSuccess){
		printf("Error\n");
		exit(0);
	}

	for(int i=0;i<n;i++){
		printf("%d \t %d \t %d\n", a[i],b[i],c[i]);
	}

	delete a;
	delete b;
	delete c;
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
