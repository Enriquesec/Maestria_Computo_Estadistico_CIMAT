#include "hip/hip_runtime.h"
#include "funciones_simulacion.h"  // archivo auxiliar con las funciones a utilizar.

int main(){
	clock_t t1, t2; // variables para el tiempo en CPU
    float tiempo_cpu, tiempo_gpu; // variables para el tiempo en GPU

    matriz h_A, h_B, h_resultado, h_resultado_secuencia; // variables tipo matriz que se ocupara en el host
    matriz dev_A, dev_B, dev_resultado; // variables tipo matriz que se ocupara en el device
	int mA, nA, mB, nB; // variables enteras para los tamanos de las matrices.
    
    // Solicitud de los tamanos de la matriz A.
    cout<<"Bienvenido al programa para operar matrices."<<endl; 
    cout<<"Ingresa el tamano de la primera matriz (A), ejemplo, " << endl;
    cout<<"para crear una matriz 5x7 ingresa 5 7 "<<endl;
    cin >> nA >> mA;
	
	// Inicializamos la matriz A en host.
	h_A.n = nA; // asignamos los tamanos a la struct matriz.  
	h_A.m = mA;
    h_A.df = new int [nA*mA]; // apartamos la memoria dinamica.
	h_A = generar_matriz_aleatoria(h_A); // generamos numeros aleatorias para la matriz.
	
	// Solicitud de los tamanos de la matriz B.
	cout<<"Ingresa el tamano de la segunda matriz (B), ejemplo, "<<endl;  
    cout << "para crear una matriz 5x7 ingresa 5 7 " << endl;
    cin >> nB >> mB; // generamos la matriz B.
    
    // Inicializamos la matriz B en host.
    h_B.n = nB; // asignamos los tamanos a la struct matriz. 
    h_B.m = mB;
    h_B.df = new int [nB*mB]; // apartamos la memoria dinamica.
    h_B = generar_matriz_aleatoria(h_B); // generamos numeros aleatorias para la matriz.
    
    dev_A.n = h_A.n; // asignamos los tamanos a la struct matriz A que estara en device
    dev_A.m = h_A.m;     
    hipMalloc(&dev_A.df, nA*mA*sizeof(int)); // asignacion de memoria lineal en device
    hipMemcpy(dev_A.df, h_A.df, nA*mA*sizeof(int), hipMemcpyHostToDevice); // copiamos los datos del host al device 

    dev_B.n = h_B.n; // asignamos los tamanos a la struct matriz B que estara en device
    dev_B.m = h_B.m;
    hipMalloc(&dev_B.df, nB*mB*sizeof(int)); // asignacion de memoria lineal en device
	hipMemcpy(dev_B.df, h_B.df, nB*mB*sizeof(int), hipMemcpyHostToDevice); // copiamos los datos del host al device

    char x, operacion, y; // variables para realizar operaciones
    cout << "Haz creado la matriz A y B. Ahora ingresa la operacion a realizar,ejemplos:" << endl;
    cout << "A+B. |A-B. |A*B. |B*A. (Nota: con el punto)" << endl; // solicitamos la operacion
    cin >> x >> operacion >> y;


    hipEvent_t start, stop; // medimos el tiempo en GPU, considerando la recomendacion la guía para CUDA
                            // ver bibliografia del reporte.
	hipEventCreate(&start); // inicializamos lo eventos
	hipEventCreate(&stop);
	hipEventRecord(start, 0); // inicio del contador

    if (x=='A' && y=='B' && operacion=='*'){
        h_resultado.n = dev_A.n;  // asignamos los tamanos a la struct matriz para los resultados de la multiplicacion en host
        h_resultado.m = dev_B.m;
        h_resultado.df = new int [dev_A.n*dev_B.m]; // apartamos la memoria dinamica para los resultados.
        
        dev_resultado.n = h_resultado.n; // asignamos los tamanos a la struct matriz para los resultados de la multiplicacion en device
        dev_resultado.m = h_resultado.m;
        hipMalloc(&dev_resultado.df, dev_A.n*dev_B.m*sizeof(int)); // asignacion de memoria lineal en device
    }
    else if (x=='B' && y=='A' && operacion=='*'){
        h_resultado.n = dev_B.n; // asignamos los tamanos a la struct matriz para los resultados de la multiplicacion en host
        h_resultado.m = dev_A.m;
        h_resultado.df = new int [dev_B.n*dev_A.m]; // apartamos la memoria dinamica para los resultados.
        
        dev_resultado.n = h_resultado.n; // asignamos los tamanos a la struct matriz para los resultados de la multiplicacion en device
        dev_resultado.m = h_resultado.m;
        hipMalloc(&dev_resultado.df, dev_B.n*dev_A.m*sizeof(int)); // asignacion de memoria lineal en device
    }
    else{
        h_resultado.n = dev_A.n; // asignamos los tamanos a la struct matriz para los resultados de la suma/resta en host
        h_resultado.m = dev_A.m;
        h_resultado.df = new int [dev_A.n*dev_A.m]; // apartamos la memoria dinamica para los resultados.
        
        dev_resultado.n = h_resultado.n; // asignamos los tamanos a la struct matriz para los resultados de la suma/resta en device
        dev_resultado.m = h_resultado.m;
        hipMalloc(&dev_resultado.df, dev_A.n*dev_A.m*sizeof(int)); // asignacion de memoria lineal en device
    }
    
    ejecutar_operacion(dev_A, dev_B, operacion, dev_resultado, h_resultado, x, y); // ejecutamos la operacion
    
    hipEventRecord(stop, 0); // detenemos el contador de tiempo

	hipEventSynchronize(stop); // esperamos a que todos los hilos terminen
	hipEventElapsedTime(&tiempo_gpu, start, stop); // tiempo de ejecucion
	hipEventDestroy(start);  // eliminamos los eventos.
	hipEventDestroy(stop);
	
	t1 = clock(); 

    //---------------Secuencia suma y resta
	// h_resultado_secuencia.n = h_A.n; // asignamos los tamanos a la struct matriz. 
    // h_resultado_secuencia.m = h_A.m;
    // h_resultado_secuencia.df = new int [h_A.n*h_A.m]; // apartamos la memoria dinamica.
    
    // suma_matriz_sec(h_A, h_B, h_resultado_secuencia);
    // resta_matriz_sec(h_A, h_B, h_resultado_secuencia);
    
    // --------------Secuencia multiplicacion
    h_resultado_secuencia.n = h_A.n; // asignamos los tamanos a la struct matriz. 
    h_resultado_secuencia.m = h_B.m;
    h_resultado_secuencia.df = new int [h_A.n*h_B.m]; // apartamos la memoria dinamica.
    
    multiplicacion_matriz_sec(h_A, h_B, h_resultado_secuencia);
    
	t2 = clock();
    tiempo_cpu = (double(t2 - t1) )/ double(CLOCKS_PER_SEC); //tiempo de ejecucion en CPU

    cout<<"\nCPU: "<<tiempo_cpu<<", GPU: "<<tiempo_gpu<<endl;

	return 0;
}